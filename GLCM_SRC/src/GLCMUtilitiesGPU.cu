#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "GLCMCalculationGPU.cuh"

void PrintGLCM(const float* const glcm, const int depth)
{
	for (int i = 0; i < depth; i++)
	{
		for (int j = 0; j < depth; j++)
		{
			printf("%20.9f ", glcm[i * depth + j]);
		}
		printf("\n");
	}
}

void CompareGLCMs(const float* const glcm1, const float* const glcm2, const int depth)
{
	printf("\nComparing GLCMs...\n");
	for (int i = 0; i < depth; i++)
	{
		for (int j = 0; j < depth; j++)
		{
			float diff = glcm1[i * depth + j] - glcm2[i * depth + j];
			if (diff != 0)
				printf("(%d,%d) DOES NOT MATCH\n", i, j);
		}
	}
	printf("...End of Comparison\n");
}

__global__ void BitmapToIntensity_kernel(const unsigned char* pixels,
	unsigned char* intensity, int size)
{
	int offset = 4;
	int uid = blockIdx.x * blockDim.x + threadIdx.x;

	if (uid < size)
	{
		unsigned char rgbBlue = pixels[uid * offset];
		unsigned char rgbGreen = pixels[uid * offset + 1];
		unsigned char rgbRed = pixels[uid * offset + 2];

		// LUMA Coding standard conversion: Y' = 0.299R' + 0.587G' + 0.114B'
		intensity[uid] = (0.299 * rgbRed) + (0.587 * rgbGreen) + (0.114 * rgbBlue);
	}
}

unsigned char* BitmapToIntensityGPU(BitmapData* bmdata)
{
	const float size = bmdata->bmi->bmiHeader.biHeight * bmdata->bmi->bmiHeader.biWidth;
	const int rgbQuadSize = 4;
	const float K = 16;

	unsigned char* d_intensity = 0;
	unsigned char* d_pixels = 0;

	dim3 blocksize(ceil(size / K), 1, 1);
	dim3 threadsize(K, 1, 1);

	checkCudaErrors(hipMalloc(&d_intensity, sizeof(unsigned char) * size));
	checkCudaErrors(hipMalloc(&d_pixels, sizeof(unsigned char) * size * rgbQuadSize));

	checkCudaErrors(hipMemcpy(d_pixels, bmdata->pixels, sizeof(unsigned char) * size * rgbQuadSize, hipMemcpyHostToDevice));

	BitmapToIntensity_kernel<<<blocksize, threadsize>>>(d_pixels, d_intensity, size);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	hipFree(d_pixels);

	return d_intensity;
}

__global__ void IntensityScaleLevels_kernel(const unsigned char* intensity,
	unsigned char* intensityNew, int size, float scaleFactor)
{
	int uid = blockIdx.x * blockDim.x + threadIdx.x;

	if (uid < size)
	{
		intensityNew[uid] = (int)((float)intensity[uid] / scaleFactor);
	}
}

unsigned char* IntensityScaleDepthGPU(const unsigned char* d_intensity, const int rows,
	const int cols, const int currentDepth, const int newDepth)
{
	const float size = rows * cols;
	const float scaleFactor = currentDepth / newDepth;
	const float K = 16;

	unsigned char* d_intensityNew = 0;

	dim3 blocksize((size + K - 1) / K, 1, 1);
	dim3 threadsize(K, 1, 1);

	checkCudaErrors(hipMalloc(&d_intensityNew, sizeof(unsigned char) * size));

	IntensityScaleLevels_kernel<<<blocksize, threadsize>>>(d_intensity, d_intensityNew, size, scaleFactor);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	return d_intensityNew;
}

void GLCMgpu_PrintFeatures(GLCMInfo &gi)
{
	printf("\nPhase 1 Features Parallel\n\n");
	printf("%30s: %.9f\n", "Mean", gi.h_ux);
	printf("%30s: %.9f\n", "Dissimilarity", gi.h_dis);
	printf("%30s: %.9f\n", "Contrast", gi.h_con);
	printf("%30s: %.9f\n", "Inverse Difference Momentum", gi.h_idm);
	printf("%30s: %.9f\n", "Entropy", gi.h_ent);
	printf("%30s: %.9f\n", "Angular Second Momentum", gi.h_asm);
	printf("%30s: %.9f\n", "Maximum Probability", gi.h_map);
	printf("%30s: %.9f\n", "Minimum Probability", gi.h_mip);
	printf("\nPhase 2 Features Parallel\n\n");
	printf("%30s: %.9f\n", "Sum Entropy", gi.h_sen);
	printf("%30s: %.9f\n", "Difference Entropy", gi.h_den);
	printf("%30s: %.9f\n", "Sum Average", gi.h_sav);
	printf("%30s: %.9f\n", "Sum Variance", gi.h_sva);
	printf("%30s: %.9f\n", "Difference Variance", gi.h_dva);
	printf("%30s: %.9f\n", "Variance", gi.h_var);
	printf("%30s: %.9f\n", "Standard Devation", gi.h_sdx);
	printf("\nPhase 3 Features Parallel\n\n");
	printf("%30s: %.9f\n", "Correlation", gi.h_cor);
	printf("%30s: %.9f\n", "Cluster Shade", gi.h_cls);
	printf("%30s: %.9f\n", "Cluster Prominance", gi.h_clp);
}

void GLCMgpu_FreeMemory(GLCMInfo &gi)
{
	hipFree(gi.d_intensity);
	hipFree(gi.d_glcm);
}

void GLCMgpu_InitializeMemory(GLCMInfo &gi, unsigned char* in_intensity)
{
	int intensitysize = sizeof(unsigned char) * gi.rows * gi.cols;
	checkCudaErrors(hipMalloc(&(gi.d_intensity), intensitysize));
	checkCudaErrors(hipMemcpy(gi.d_intensity, in_intensity, intensitysize, hipMemcpyHostToDevice));

	int glcmsize = sizeof(float) * gi.depth * gi.depth;
	checkCudaErrors(hipMalloc(&(gi.d_glcm), glcmsize));
	checkCudaErrors(hipMemset(gi.d_glcm, 0, glcmsize));
}

void GLCMgpu_GetGLCMMatrixCPU(GLCMInfo &gi, float*& glcm)
{
	checkCudaErrors(hipMemcpy(glcm, gi.d_glcm, sizeof(float) * gi.depth * gi.depth, hipMemcpyDeviceToHost));
}