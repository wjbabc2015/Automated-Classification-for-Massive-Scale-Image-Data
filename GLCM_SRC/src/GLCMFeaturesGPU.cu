#include "hip/hip_runtime.h"
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "GLCMCalculationGPU.cuh"

template <unsigned int blocksize>
__device__ void MinWarpReduce(volatile float* shared, unsigned int tid) {
	if (blocksize >= 64) shared[tid] = fminf(shared[tid], shared[tid + 32]);
	if (blocksize >= 32) shared[tid] = fminf(shared[tid], shared[tid + 16]);
	if (blocksize >= 16) shared[tid] = fminf(shared[tid], shared[tid + 8]);
	if (blocksize >= 8)  shared[tid] = fminf(shared[tid], shared[tid + 4]);
	if (blocksize >= 4)  shared[tid] = fminf(shared[tid], shared[tid + 2]);
	if (blocksize >= 2)  shared[tid] = fminf(shared[tid], shared[tid + 1]);
}

template<unsigned int blocksize>
__device__ void MinReduction_dev_kernel(float* shared)
{
	int tid = threadIdx.x;

	if (blocksize >= 256)
	{
		if (tid < 128)
			shared[tid] = fminf(shared[tid], shared[tid + 128]);
		__syncthreads();
	}

	if (blocksize >= 128)
	{
		if (tid < 64)
			shared[tid] = fminf(shared[tid], shared[tid + 64]);
		__syncthreads();
	}

	if (tid < 32) MinWarpReduce<blocksize>(shared, tid);

	//if (tid < 32)
	//{
	//	if (blocksize >= 64)
	//		shared[tid] = fminf(shared[tid], shared[tid + 32]);
	//	if (blocksize >= 32)
	//		shared[tid] = fminf(shared[tid], shared[tid + 16]);
	//	if (blocksize >= 16)
	//		shared[tid] = fminf(shared[tid], shared[tid + 8]);
	//	if (blocksize >= 8)
	//		shared[tid] = fminf(shared[tid], shared[tid + 4]);
	//	if (blocksize >= 4)
	//		shared[tid] = fminf(shared[tid], shared[tid + 2]);
	//	if (blocksize >= 2)
	//		shared[tid] = fminf(shared[tid], shared[tid + 1]);
	//}
}

template <unsigned int blocksize>
__device__ void MaxWarpReduce(volatile float* shared, unsigned int tid) {
	if (blocksize >= 64) shared[tid] = fmaxf(shared[tid], shared[tid + 32]);
	if (blocksize >= 32) shared[tid] = fmaxf(shared[tid], shared[tid + 16]);
	if (blocksize >= 16) shared[tid] = fmaxf(shared[tid], shared[tid + 8]);
	if (blocksize >= 8)  shared[tid] = fmaxf(shared[tid], shared[tid + 4]);
	if (blocksize >= 4)  shared[tid] = fmaxf(shared[tid], shared[tid + 2]);
	if (blocksize >= 2)  shared[tid] = fmaxf(shared[tid], shared[tid + 1]);
} 

template<unsigned int blocksize>
__device__ void MaxReduction_dev_kernel(float* shared)
{
	int tid = threadIdx.x;

	if (blocksize >= 256)
	{
		if (tid < 128)
			shared[tid] = fmaxf(shared[tid], shared[tid + 128]);
		__syncthreads();
	}

	if (blocksize >= 128)
	{
		if (tid < 64)
			shared[tid] = fmaxf(shared[tid], shared[tid + 64]);
		__syncthreads();
	}

	if (tid < 32) MaxWarpReduce<blocksize>(shared, tid);

	//if (tid < 32)
	//{
	//	if (blocksize >= 64)
	//		shared[tid] = fmaxf(shared[tid], shared[tid + 32]);
	//	if (blocksize >= 32)
	//		shared[tid] = fmaxf(shared[tid], shared[tid + 16]);
	//	if (blocksize >= 16)
	//		shared[tid] = fmaxf(shared[tid], shared[tid + 8]);
	//	if (blocksize >= 8)
	//		shared[tid] = fmaxf(shared[tid], shared[tid + 4]);
	//	if (blocksize >= 4)
	//		shared[tid] = fmaxf(shared[tid], shared[tid + 2]);
	//	if (blocksize >= 2)
	//		shared[tid] = fmaxf(shared[tid], shared[tid + 1]);
	//}
}

template <unsigned int blocksize>
__device__ void SumWarpReduce(volatile float* shared, unsigned int tid) {
	if (blocksize >= 64) shared[tid] += shared[tid + 32];
	if (blocksize >= 32) shared[tid] += shared[tid + 16];
	if (blocksize >= 16) shared[tid] += shared[tid + 8];
	if (blocksize >= 8)  shared[tid] += shared[tid + 4];
	if (blocksize >= 4)  shared[tid] += shared[tid + 2];
	if (blocksize >= 2)  shared[tid] += shared[tid + 1];
}

template<unsigned int blocksize>
__device__ void SumReduction_dev_kernel(float* shared)
{
	int tid = threadIdx.x;

	if (blocksize >= 256)
	{
		if (tid < 128)
			shared[tid] += shared[tid + 128];
		__syncthreads();
	}

	if (blocksize >= 128)
	{
		if (tid < 64)
			shared[tid] += shared[tid + 64];
		__syncthreads();
	}

	if (tid < 32) SumWarpReduce<blocksize>(shared, tid);

	//if (tid < 32)
	//{
	//	if (blocksize >= 64)
	//		shared[tid] += shared[tid + 32];
	//	if (blocksize >= 32)
	//		shared[tid] += shared[tid + 16];
	//	if (blocksize >= 16)
	//		shared[tid] += shared[tid + 8];
	//	if (blocksize >= 8)
	//		shared[tid] += shared[tid + 4];
	//	if (blocksize >= 4)
	//		shared[tid] += shared[tid + 2];
	//	if (blocksize >= 2)
	//		shared[tid] += shared[tid + 1];
	//}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP1A_kernel(float* g_glcm, float* g_ux, float* g_dis,
	float* g_con, float* g_idm, float* g_ent, float* g_asm, float* g_map, float* g_mip)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int uid = blockIdx.x * (blocksize * 2) + tid;
	int i = blockIdx.x * 2;
	int j = threadIdx.x;

	float a = g_glcm[uid];
	float b = g_glcm[uid + blocksize];

	// mean-x
	{
		shared[tid] = (i * a) + ((i + 1) * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_ux[blockIdx.x] = shared[0];
	}

	// dissimilarity
	{
		shared[tid] = (a * abs(i - j)) + (b * abs((i + 1) - j));
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_dis[blockIdx.x] = shared[0];
	}

	// contrast
	{
		int at = i - j;
		int bt = i + 1 - j;
		shared[tid] = (a * (at * at)) + (b * (bt * bt));
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_con[blockIdx.x] = shared[0];
	}

	// inverse difference momentum
	{
		float at = (float)(i - j);
		float bt = (float)(i + 1 - j);
		shared[tid] = (a * (1.0f / (1.0f + (at * at)))) + (b * (1.0f / (1.0f + (bt * bt))));
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_idm[blockIdx.x] = shared[0];
	}

	// entropy
	{
		float loga = (a == 0) ? (0.0f) : (log(a));
		float logb = (b == 0) ? (0.0f) : (log(b));
		shared[tid] = (a * loga) + (b * logb);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_ent[blockIdx.x] = shared[0];
	}

	// angular second momentum
	{
		shared[tid] = (a * a) + (b * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_asm[blockIdx.x] = shared[0];
	}

	// max probability
	{
		shared[tid] = fmaxf(a, b);
		__syncthreads();
		MaxReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_map[blockIdx.x] = shared[0];
	}

	// min probability
	{
		shared[tid] = fminf(a, b);
		__syncthreads();
		MinReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_mip[blockIdx.x] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP1B_kernel(float* g_ux, float* g_dis,
	float* g_con, float* g_idm, float* g_ent, float* g_asm, float* g_map, float* g_mip)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int uid = blockIdx.x * (blocksize * 2) + tid;

	// mean-x
	{
		shared[tid] = g_ux[uid] + g_ux[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_ux[blockIdx.x] = shared[0];
	}

	// dissimilarity
	{
		shared[tid] = g_dis[uid] + g_dis[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_dis[blockIdx.x] = shared[0];
	}

	// contrast
	{
		shared[tid] = g_con[uid] + g_con[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_con[blockIdx.x] = shared[0];
	}

	// inverse difference momentum
	{
		shared[tid] = g_idm[uid] + g_idm[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_idm[blockIdx.x] = shared[0];
	}

	// entropy
	{
		shared[tid] = g_ent[uid] + g_ent[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_ent[blockIdx.x] = -shared[0];
	}

	// angular second momentum
	{
		shared[tid] = g_asm[uid] + g_asm[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_asm[blockIdx.x] = shared[0];
	}

	// max probability
	{
		shared[tid] = fmaxf(g_map[uid], g_map[uid + blocksize]);
		__syncthreads();
		MaxReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_map[blockIdx.x] = shared[0];
	}

	// min probability
	{
		shared[tid] = fminf(g_mip[uid], g_mip[uid + blocksize]);
		__syncthreads();
		MinReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_mip[blockIdx.x] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP1C_kernel(float* g_glcm, float* g_pxpy)
{
	extern __shared__ float shared[];

	int k = blockIdx.x;
	int i = threadIdx.x;

	// anti diagonal probability
	{
		int j = k - i;
		shared[i] = (j < 0 || j >= blocksize) ? 0.0f : g_glcm[i * blocksize + j];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (i == 0) g_pxpy[k] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP1D_kernel(float* g_glcm, float* g_pxmy)
{
	extern __shared__ float shared[];

	int k = blockIdx.x;
	int i = threadIdx.x;

	// main diagonal probability
	{
		int j = k + i;
		shared[i] = (j >= blocksize) ? 0.0f : g_glcm[i * blocksize + j];
		if (i != j) shared[i] += shared[i];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (i == 0) g_pxmy[k] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP2A_kernel(float* g_pxpy, float* g_sen, float* g_sav, float* g_sva)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int k = threadIdx.x * 2;

	float a = g_pxpy[k];
	float b = g_pxpy[k + 1];
	float sen;

	// sum entropy
	{
		float loga = (a == 0) ? (0.0f) : (log(a));
		float logb = (b == 0) ? (0.0f) : (log(b));
		shared[tid] = (a * loga) + (b * logb);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_sen[blockIdx.x] = -shared[0];
	}

	__syncthreads();

	// sum variance
	{
		sen = shared[0];
		float kdif = (float)k - sen;
		float k2dif = (float)(k + 1) - sen;
		shared[tid] = ((kdif * kdif) * a) + ((k2dif * k2dif) * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_sva[blockIdx.x] = shared[0];
	}

	// sum average
	{
		shared[tid] = ((float)k * a) + ((float)(k + 1) * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_sav[blockIdx.x] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP2B_kernel(float* g_pxmy, float* g_den, float* g_dva)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int k = threadIdx.x * 2;

	float a = g_pxmy[k];
	float b = g_pxmy[k + 1];

	// difference entropy
	{
		float loga = (a == 0) ? (0.0f) : (log(a));
		float logb = (b == 0) ? (0.0f) : (log(b));
		shared[tid] = (a * loga) + (b * logb);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_den[blockIdx.x] = -shared[0];
	}

	// difference variance
	{
		shared[tid] = ((float)(k * k) * a) + ((float)((k + 1) * (k + 1)) * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_dva[blockIdx.x] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP2C_kernel(float* g_glcm, float* g_ux, float* g_var)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int uid = blockIdx.x * (blocksize * 2) + tid;
	int i = blockIdx.x * 2;

	float a = g_glcm[uid];
	float b = g_glcm[uid + blocksize];

	// variance
	{
		float ux = *g_ux;
		float adif = ((float)i - ux);
		float bdif = ((float)(i + 1) - ux);
		shared[tid] = ((adif * adif) * a) + ((bdif * bdif) * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_var[blockIdx.x] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP2D_kernel(float* g_var, float* g_sdx)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int uid = blockIdx.x * (blocksize * 2) + tid;

	// variance & standard deviation
	{
		shared[tid] = g_var[uid] + g_var[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0)
		{
			g_var[blockIdx.x] = shared[0];
			g_sdx[blockIdx.x] = sqrtf(shared[0]);
		}
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP3A_kernel(float* g_glcm, float* g_ux, float* g_cor, float* g_cls, float* g_clp)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int uid = blockIdx.x * (blocksize * 2) + tid;
	float i = blockIdx.x * 2;
	float j = threadIdx.x;

	float a = g_glcm[uid];
	float b = g_glcm[uid + blocksize];
	float ux = *g_ux;

	// correlation
	{
		float jminus_ux = j - ux;
		shared[tid] = (((i - ux) * jminus_ux) * a) + ((((i + 1.0f) - ux) * jminus_ux) * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_cor[blockIdx.x] = shared[0];
	}

	float ijdif = (i + j) - ux - ux;
	float ijdif_cubed = ijdif * ijdif * ijdif;
	float i2jdif = ((i + 1.0f) + j) - ux - ux;
	float i2jdif_cubed = i2jdif * i2jdif * i2jdif;

	// cluster shade
	{
		shared[tid] = (ijdif_cubed * a) + (i2jdif_cubed * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_cls[blockIdx.x] = shared[0];
	}

	// cluster prominence
	{
		shared[tid] = ((ijdif_cubed * ijdif) * a) + ((i2jdif_cubed * i2jdif) * b);
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_clp[blockIdx.x] = shared[0];
	}
}

template<unsigned int blocksize>
__global__ void GLCMFeaturesP3B_kernel(float* g_cor, float* g_var, float* g_cls, float* g_clp)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int uid = blockIdx.x * (blocksize * 2) + tid;

	// correlation
	{
		float var = *g_var;
		if (var == 0)
		{
			g_cor[0] = 1.0f;
		}
		else
		{
			shared[tid] = g_cor[uid] + g_cor[uid + blocksize];
			__syncthreads();
			SumReduction_dev_kernel<blocksize>(shared);
			if (tid == 0) g_cor[blockIdx.x] = shared[0] / sqrtf(var * var);
		}
	}

	// cluster shade
	{
		shared[tid] = g_cls[uid] + g_cls[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_cls[blockIdx.x] = shared[0];
	}

	// cluster prominence
	{
		shared[tid] = g_clp[uid] + g_clp[uid + blocksize];
		__syncthreads();
		SumReduction_dev_kernel<blocksize>(shared);
		if (tid == 0) g_clp[blockIdx.x] = shared[0];
	}
}


void GLCMgpu_CalculateFeatures(GLCMInfo &gi, float* time)
{
	int depth = gi.depth;
	float* d_glcm = gi.d_glcm;

	int diagRange = 2 * depth - 1;

	float* d_ux = 0;
	float* d_dis = 0;
	float* d_con = 0;
	float* d_idm = 0;
	float* d_ent = 0;
	float* d_asm = 0;
	float* d_map = 0;
	float* d_mip = 0;
	float* d_pxpy = 0;
	float* d_pxmy = 0;

	float* d_sen = 0;
	float* d_sav = 0;
	float* d_sva = 0;
	float* d_den = 0;
	float* d_dva = 0;
	float* d_var = 0;
	float* d_sdx = 0;

	float* d_cor = 0;
	float* d_cls = 0;
	float* d_clp = 0;

	float time1, time2, time3;

	hipEvent_t start = 0, stop = 0;

	checkCudaErrors(hipEventCreate(&start, 0));
	checkCudaErrors(hipEventCreate(&stop, 0));

	checkCudaErrors(hipMalloc(&d_ux, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_dis, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_con, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_idm, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_ent, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_asm, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_map, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_mip, sizeof(float) * (depth/2)));
	checkCudaErrors(hipMalloc(&d_pxpy, sizeof(float) * (depth*2)));
	checkCudaErrors(hipMemset(d_pxpy, 0.0f, (depth*2)));
	checkCudaErrors(hipMalloc(&d_pxmy, sizeof(float) * depth));

	checkCudaErrors(hipMalloc(&d_sen, sizeof(float)* diagRange));
	checkCudaErrors(hipMalloc(&d_sav, sizeof(float)* depth));
	checkCudaErrors(hipMalloc(&d_sva, sizeof(float)* (depth/2)));
	checkCudaErrors(hipMalloc(&d_den, sizeof(float)* (depth/2)));
	checkCudaErrors(hipMalloc(&d_dva, sizeof(float)* (depth/2)));
	checkCudaErrors(hipMalloc(&d_var, sizeof(float)* (depth/2)));
	checkCudaErrors(hipMalloc(&d_sdx, sizeof(float)));

	checkCudaErrors(hipMalloc(&d_cor, sizeof(float)* (depth/2)));
	checkCudaErrors(hipMalloc(&d_cls, sizeof(float)* (depth/2)));
	checkCudaErrors(hipMalloc(&d_clp, sizeof(float)* (depth/2)));


	switch (depth)
	{
	case 256:
		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP1A_kernel<256><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1B_kernel<64><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1C_kernel<256><<<diagRange, depth, sizeof(float) * depth>>>(d_glcm, d_pxpy);
		GLCMFeaturesP1D_kernel<256><<<depth, depth, sizeof(float) * depth>>>(d_glcm, d_pxmy);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time1, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP2A_kernel<256><<<1, depth, sizeof(float) * depth>>>(d_pxpy, d_sen, d_sav, d_sva);
		GLCMFeaturesP2B_kernel<128><<<1, depth/2, sizeof(float) * (depth/2)>>>(d_pxmy, d_den, d_dva);
		GLCMFeaturesP2C_kernel<256><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_var);
		GLCMFeaturesP2D_kernel<64><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_var, d_sdx);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time2, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP3A_kernel<256><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_cor, d_cls, d_clp);
		GLCMFeaturesP3B_kernel<64><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_cor, d_var, d_cls, d_clp);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time3, start, stop));

		break;
	case 128:
		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP1A_kernel<128><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1B_kernel<32><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1C_kernel<128><<<diagRange, depth, sizeof(float) * depth>>>(d_glcm, d_pxpy);
		GLCMFeaturesP1D_kernel<128><<<depth, depth, sizeof(float) * depth>>>(d_glcm, d_pxmy);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time1, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP2A_kernel<128><<<1, depth, sizeof(float) * depth>>>(d_pxpy, d_sen, d_sav, d_sva);
		GLCMFeaturesP2B_kernel<64><<<1, depth/2, sizeof(float) * (depth/2)>>>(d_pxmy, d_den, d_dva);
		GLCMFeaturesP2C_kernel<128><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_var);
		GLCMFeaturesP2D_kernel<32><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_var, d_sdx);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time2, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP3A_kernel<128><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_cor, d_cls, d_clp);
		GLCMFeaturesP3B_kernel<32><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_cor, d_var, d_cls, d_clp);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time3, start, stop));

		break;
	case 64:
		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP1A_kernel<64><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1B_kernel<16><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1C_kernel<64><<<diagRange, depth, sizeof(float) * depth>>>(d_glcm, d_pxpy);
		GLCMFeaturesP1D_kernel<64><<<depth, depth, sizeof(float) * depth>>>(d_glcm, d_pxmy);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time1, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP2A_kernel<64><<<1, depth, sizeof(float) * depth>>>(d_pxpy, d_sen, d_sav, d_sva);
		GLCMFeaturesP2B_kernel<32><<<1, depth/2, sizeof(float) * (depth/2)>>>(d_pxmy, d_den, d_dva);
		GLCMFeaturesP2C_kernel<64><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_var);
		GLCMFeaturesP2D_kernel<16><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_var, d_sdx);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time2, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP3A_kernel<64><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_cor, d_cls, d_clp);
		GLCMFeaturesP3B_kernel<16><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_cor, d_var, d_cls, d_clp);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time3, start, stop));

		break;
	case 32:
		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP1A_kernel<32><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1B_kernel<8><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1C_kernel<32><<<diagRange, depth, sizeof(float) * depth>>>(d_glcm, d_pxpy);
		GLCMFeaturesP1D_kernel<32><<<depth, depth, sizeof(float) * depth>>>(d_glcm, d_pxmy);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time1, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP2A_kernel<32><<<1, depth, sizeof(float) * depth>>>(d_pxpy, d_sen, d_sav, d_sva);
		GLCMFeaturesP2B_kernel<16><<<1, depth/2, sizeof(float) * (depth/2)>>>(d_pxmy, d_den, d_dva);
		GLCMFeaturesP2C_kernel<32><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_var);
		GLCMFeaturesP2D_kernel<8><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_var, d_sdx);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time2, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP3A_kernel<32><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_cor, d_cls, d_clp);
		GLCMFeaturesP3B_kernel<8><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_cor, d_var, d_cls, d_clp);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time3, start, stop));

		break;
	case 16:
		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP1A_kernel<16><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1B_kernel<4><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1C_kernel<16><<<diagRange, depth, sizeof(float) * depth>>>(d_glcm, d_pxpy);
		GLCMFeaturesP1D_kernel<16><<<depth, depth, sizeof(float) * depth>>>(d_glcm, d_pxmy);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time1, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP2A_kernel<16><<<1, depth, sizeof(float) * depth>>>(d_pxpy, d_sen, d_sav, d_sva);
		GLCMFeaturesP2B_kernel<8><<<1, depth/2, sizeof(float) * (depth/2)>>>(d_pxmy, d_den, d_dva);
		GLCMFeaturesP2C_kernel<16><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_var);
		GLCMFeaturesP2D_kernel<4><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_var, d_sdx);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time2, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP3A_kernel<16><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_cor, d_cls, d_clp);
		GLCMFeaturesP3B_kernel<4><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_cor, d_var, d_cls, d_clp);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time3, start, stop));

		break;
	case 8:
		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP1A_kernel<8><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1B_kernel<2><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_ux, d_dis, d_con, d_idm, d_ent, d_asm, d_map, d_mip);
		GLCMFeaturesP1C_kernel<8><<<diagRange, depth, sizeof(float) * depth>>>(d_glcm, d_pxpy);
		GLCMFeaturesP1D_kernel<8><<<depth, depth, sizeof(float) * depth>>>(d_glcm, d_pxmy);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time1, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP2A_kernel<8><<<1, depth, sizeof(float) * depth>>>(d_pxpy, d_sen, d_sav, d_sva);
		GLCMFeaturesP2B_kernel<4><<<1, depth/2, sizeof(float) * (depth/2)>>>(d_pxmy, d_den, d_dva);
		GLCMFeaturesP2C_kernel<8><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_var);
		GLCMFeaturesP2D_kernel<2><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_var, d_sdx);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time2, start, stop));

		checkCudaErrors(hipEventRecord(start, 0));

		GLCMFeaturesP3A_kernel<8><<<depth/2, depth, sizeof(float) * depth>>>(d_glcm, d_ux, d_cor, d_cls, d_clp);
		GLCMFeaturesP3B_kernel<2><<<1, depth/4, sizeof(float) * (depth/4)>>>(d_cor, d_var, d_cls, d_clp);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&time3, start, stop));

		break;
	default:
		// handle?
		break;
	}

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	printf("time elapsed in p1: %f\n", time1);
	printf("time elapsed in p2: %f\n", time2);
	printf("time elapsed in p2: %f\n", time3);

	checkCudaErrors(hipMemcpy(&(gi.h_ux), &d_ux[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_dis), &d_dis[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_con), &d_con[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_idm), &d_idm[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_ent), &d_ent[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_asm), &d_asm[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_map), &d_map[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_mip), &d_mip[0], sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(&(gi.h_sen), &d_sen[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_sav), &d_sav[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_sva), &d_sva[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_den), &d_den[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_dva), &d_dva[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_var), &d_var[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_sdx), &d_sdx[0], sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(&(gi.h_cor), &d_cor[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_cls), &d_cls[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&(gi.h_clp), &d_clp[0], sizeof(float), hipMemcpyDeviceToHost));

	hipFree(d_ux);
	hipFree(d_dis);
	hipFree(d_con);
	hipFree(d_idm);
	hipFree(d_ent);
	hipFree(d_asm);
	hipFree(d_map);
	hipFree(d_mip);
	hipFree(d_pxpy);
	hipFree(d_pxmy);

	hipFree(d_sen);
	hipFree(d_sav);
	hipFree(d_sva);
	hipFree(d_den);
	hipFree(d_dva);
	hipFree(d_var);
	hipFree(d_sdx);

	hipFree(d_cor);
	hipFree(d_cls);
	hipFree(d_clp);

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
}