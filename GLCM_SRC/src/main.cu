#include <Windows.h>
#include <WinUser.h>
#include <stdio.h>
#include <malloc.h>
#include <math.h>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "GLCMCalculationCPU.h"
#include "GLCMCalculationGPU.cuh"
#include "BitmapLoader.h"

#include "HRTimer.h"

const LPCSTR IMAGE = "bm1.bmp";

//int main(int argc, char** argv)
//{
//	BitmapData* bmdata = LoadBitmapData(IMAGE);
//	unsigned char* d_intensity = BitmapToIntensityGPU(bmdata);
//	int rows = bmdata->bmi->bmiHeader.biHeight;
//	int cols = bmdata->bmi->bmiHeader.biWidth;
//
//	unsigned char* d_intensityD8 = IntensityScaleDepthGPU(d_intensity, rows, cols, 256, 8);
//
//	//float* glcmNorm;
//	//GLCMGetNormalizedMatrix(glcminfo, glcmNorm);
//	//PrintGLCM(glcmNorm, glcminfo->depth);
//	//free(glcmNorm);
//
//	for (int i = 0; i < 10; i++) 
//	{
//		GLCMInfo* glcminfo = new GLCMInfo(rows, cols, 256, 0, 1);
//		GLCMInitializeMemory(glcminfo, d_intensity);
//		GLCMCalculateGPU(glcminfo);
//
//		float time;
//		GLCMCalculateFeaturesGPU(glcminfo, &time);
//		GLCMPrintFeatures(glcminfo);
//
//		//GLCMFreeGPUMemory(glcminfo);
//	}
//
//	hipFree(d_intensity);
//	hipFree(d_intensityD8);
//}

void ComputeOnCPU(unsigned char* intensity, int rows, int cols, int depth)
{
	stopWatch timer;
	GLCMInfoCPU c_gi(rows, cols, depth, 0, 1);
	GLCMcpu_InitializeMemory(c_gi, intensity);
	GLCMcpu_CalculateGLCM(c_gi);
	GLCMcpu_NormalizeGLCM(c_gi);
	GLCMcpu_CalculateFeatures(c_gi);
	printf("\n(CPU) Features for Depth %d\n", depth);
	GLCMcpu_PrintFeatureSet(c_gi.F);
	GLCMcpu_FreeMemory(c_gi);	
	printf("\n ************************ \n");
}

void ComputeOnGPU(unsigned char* intensity, int rows, int cols, int depth)
{
	float timer;
	GLCMInfo g_gi(rows, cols, depth, 0, 1);
	GLCMgpu_InitializeMemory(g_gi, intensity);
	GLCMgpu_CalculateGLCM(g_gi);
	GLCMgpu_NormalizeGLCM(g_gi);
	GLCMgpu_CalculateFeatures(g_gi, &timer);
	printf("\n(GPU) Features for Depth %d\n", depth);
	GLCMgpu_PrintFeatures(g_gi);
	GLCMgpu_FreeMemory(g_gi);
	printf("\n ---- \n");
}

int main(int argc, char** argv)
{
	BitmapData* bmdata = LoadBitmapData(IMAGE);
	int rows = bmdata->bmi->bmiHeader.biHeight;
	int cols = bmdata->bmi->bmiHeader.biWidth;

	printf("\nwidth: %d\nheight: %d\nbits per pixel: %d\n\n", bmdata->bmi->bmiHeader.biHeight, bmdata->bmi->bmiHeader.biWidth, bmdata->bmi->bmiHeader.biBitCount);

	unsigned char* intensityD256 = BitmapToIntensityCPU(bmdata);
	unsigned char* intensityD128 = IntensityScaleDepthCPU(intensityD256, rows * cols, 256, 128);
	unsigned char* intensityD64  = IntensityScaleDepthCPU(intensityD256, rows * cols, 256, 64);
	unsigned char* intensityD32  = IntensityScaleDepthCPU(intensityD256, rows * cols, 256, 32);
	unsigned char* intensityD16  = IntensityScaleDepthCPU(intensityD256, rows * cols, 256, 16);
	unsigned char* intensityD8   = IntensityScaleDepthCPU(intensityD256, rows * cols, 256, 8);

	ComputeOnGPU(intensityD256, rows, cols, 256);
	ComputeOnCPU(intensityD256, rows, cols, 256);
	ComputeOnGPU(intensityD128, rows, cols, 128);
	ComputeOnCPU(intensityD128, rows, cols, 128);
	ComputeOnGPU(intensityD64, rows, cols, 64);
	ComputeOnCPU(intensityD64, rows, cols, 64);
	ComputeOnGPU(intensityD32, rows, cols, 32);
	ComputeOnCPU(intensityD32, rows, cols, 32);
	ComputeOnGPU(intensityD16, rows, cols, 16);
	ComputeOnCPU(intensityD16, rows, cols, 16);
	ComputeOnGPU(intensityD8, rows, cols, 8);
	ComputeOnCPU(intensityD8, rows, cols, 8);

	free(intensityD256);
	free(intensityD128);
	free(intensityD64);
	free(intensityD32);
	free(intensityD16);
	free(intensityD8);
	delete(bmdata);

	return 0;
}