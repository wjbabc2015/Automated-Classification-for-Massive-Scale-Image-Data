#include "hip/hip_runtime.h"
#include <malloc.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "GLCMCalculationGPU.cuh"

__global__ void GLCMNormalize_kernel(float* glcm, const int R)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	glcm[id] /= R;
}

void GLCMgpu_NormalizeGLCM(GLCMInfo &gi)
{
	GLCMNormalize_kernel<<<gi.depth, gi.depth>>>(gi.d_glcm, gi.R);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}

__global__ void GLCMPerGrid_kernel(const unsigned char* intensity, float* glcm, const int rows, const int cols,
	const int depth, const int xmin, const int xmax, const int ymin, const int copixel)
{
	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int iid = x + y * cols;

	if (x >= xmin && x < xmax && y >= ymin && y < rows)
	{
		unsigned char i = intensity[iid];
		unsigned char j = intensity[iid + copixel];
		atomicAdd(&(glcm[i * depth + j]), 1);
		atomicAdd(&(glcm[j * depth + i]), 1);
	}
}

void GLCMPerGrid(GLCMInfo &gi, float* time)
{
	const int K = 16;

	int xmin = 0, xmax = 0, ymin = 0, copixel = 0;

	if		(gi.angle == 0)   { xmin = 0;           xmax = gi.cols - gi.distance; ymin = 0;			  copixel = 1;			    }
	else if (gi.angle == 45)  { xmin = 0;		    xmax = gi.cols - gi.distance; ymin = gi.distance; copixel = 1 - gi.cols;  }
	else if (gi.angle == 90)  { xmin = 0;			xmax = gi.cols;               ymin = gi.distance; copixel = -(gi.cols);   }
	else if (gi.angle == 135) { xmin = gi.distance; xmax = gi.cols;               ymin = gi.distance; copixel = -1 - gi.cols; }
	else {} // invalid angle -- handle?

	dim3 blocks((gi.cols + K - 1) / K, (gi.rows + K - 1) / K);
	dim3 threads(K, K);
	hipEvent_t start = 0, stop = 0;

	checkCudaErrors(hipEventCreate(&start, 0));
	checkCudaErrors(hipEventCreate(&stop, 0));

	checkCudaErrors(hipEventRecord(start, 0));
	GLCMPerGrid_kernel<<<blocks, threads>>>(gi.d_intensity, gi.d_glcm, gi.rows, gi.cols, gi.depth, xmin, xmax, ymin, copixel);
	checkCudaErrors(hipEventRecord(stop, 0));
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventElapsedTime(time, start, stop));

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

__global__ void GLCMPerBlock_kernel(const unsigned char* intensity, float* glcm, const int rows, const int cols,
	const int depth, const int xmin, const int xmax, const int ymin, const int copixel)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int iid = x + y * cols;
	int tid = threadIdx.x + threadIdx.y * blockDim.x;
	int blocksize = blockDim.x * blockDim.y;
	int glcmsize = depth * depth;

	extern __shared__ int glcmshared[];
	int k = tid;
	while (k < glcmsize)
	{
		glcmshared[k] = 0;
		k += blocksize;
	}

	__syncthreads();

	if (x >= xmin && x < xmax && y >= ymin && y < rows)
	{
		unsigned char i = intensity[iid];
		unsigned char j = intensity[iid + copixel];
		atomicAdd(&(glcmshared[i * depth + j]), 1);
		atomicAdd(&(glcmshared[j * depth + i]), 1);
	}

	__syncthreads();

	k = tid;
	while (k < glcmsize)
	{
		atomicAdd(&(glcm[k]), glcmshared[k]);
		k += blocksize;
	}
}

void GLCMPerBlock(GLCMInfo &gi, float* time)
{
	const int glcmsize = gi.depth * gi.depth * sizeof(float);
	const int K = 16;

	int xmin = 0, xmax = 0, ymin = 0, copixel = 0;

	if		(gi.angle == 0)   { xmin = 0;            xmax = gi.cols - gi.distance; ymin = 0;			   copixel = 1;             }
	else if (gi.angle == 45)  { xmin = 0;		      xmax = gi.cols - gi.distance; ymin = gi.distance; copixel = 1 - gi.cols;  }
	else if (gi.angle == 90)  { xmin = 0;			  xmax = gi.cols;                ymin = gi.distance; copixel = -(gi.cols);   }
	else if (gi.angle == 135) { xmin = gi.distance; xmax = gi.cols;                ymin = gi.distance; copixel = -1 - gi.cols; }
	else {} // invalid angle -- handle?

	dim3 blocks((gi.cols + K - 1) / K, (gi.rows + K - 1) / K);
	dim3 threads(K, K);
	hipEvent_t start = 0, stop = 0;

	checkCudaErrors(hipEventCreate(&start, 0));
	checkCudaErrors(hipEventCreate(&stop, 0));

	checkCudaErrors(hipEventRecord(start, 0));
	GLCMPerBlock_kernel<<<blocks, threads, glcmsize>>>(gi.d_intensity, gi.d_glcm, gi.rows, gi.cols, gi.depth, xmin, xmax, ymin, copixel);
	checkCudaErrors(hipEventRecord(stop, 0));
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventElapsedTime(time, start, stop));

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

void GLCMgpu_CalculateGLCM(GLCMInfo &gi)
{
	float timeElapsed;

	if (gi.depth > 64)
		GLCMPerGrid(gi, &timeElapsed);
	else
		GLCMPerBlock(gi, &timeElapsed);
}